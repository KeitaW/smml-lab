
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello from GPU thread %d!\n", threadIdx.x);
}

int main(int argc, char **argv) {
    printf("Hello, CUDA!\n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceReset(); // If you comment out this line, you will not see the line printed by the GPU
    // cudaDeviceSynchronize(); // If you use cudaDeviceSynchronize() instead of cudaDeviceReset(), you will see all the lines printed by the GPU appears at once
    return 0;
}